#include "hip/hip_runtime.h"
	#include "GPUDistLib_internal.h"

#if	IS_COMP_MULTI_POINTS
__constant__ float4	pf4Points_const[MAX_NR_OF_COMP_POINTS];
#endif	// #if	IS_COMP_MULTI_POINTS

#include "kernel_CompDist.h"	
#include "kernel_CompDistFromTriangle.h"	
#include "kernel_TransformTriangle.h"	

bool bIsUsingCpu;
bool bIsPrintingTiming;

__constant__ int	iDummy_const;

#if	defined(WITH_CUDPP)
CUDPPHandle cudpp;
#endif	// #if	defined(WITH_CUDPP)

bool
BGPUDistIsDistSquaredRoot
(
		)
{
	#if	IS_SQRT
	return true;
	#else	// #if	IS_SQRT
	return false;
	#endif	// #if	IS_SQRT
}

void
_GPUDistFree()
{
	#if	defined(WITH_CUDPP)
	ASSERT_OR_LOG(
		CUDPP_SUCCESS == cudppDestroy(cudpp),
		"");
	#endif	// #if	defined(WITH_CUDPP)
}

void
_GPUDistInit()
{
	int iDummy = 0;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMemcpyToSymbol(HIP_SYMBOL(iDummy_const),	&iDummy,	sizeof(iDummy),	0, hipMemcpyHostToDevice) );

	#if	defined(WITH_CUDPP)
	ASSERT_OR_LOG(
		CUDPP_SUCCESS == cudppCreate(&cudpp),
		"");
	#endif	// #if	defined(WITH_CUDPP)
	atexit(_GPUDistFree);
}

void
_GPUDistUseCpu
(
		bool bIsUsingCpu
		)
{
	::bIsUsingCpu = bIsUsingCpu;
}

void
_GPUDistPrintTiming
(
		bool bIsPrintingTiming
		)
{
	::bIsPrintingTiming = bIsPrintingTiming;
}

void
_GPUDistComputeDistanceFieldFromPoints
(
	size_t uNrOfPoints,
	float4 pf4Points[],
	size_t uWidth,
	size_t uHeight,
	size_t uDepth,
	float pfDist[]
)
{
	if( bIsUsingCpu )
	{
		_GPUDistComputeDistanceFieldFromPointsByCpu
			(
				uNrOfPoints,
				pf4Points,
				uWidth,
				uHeight,
				uDepth,
				pfDist
			);

		return;
	}
LIBCLOCK_INIT(bIsPrintingTiming, __FUNCTION__);
LIBCLOCK_BEGIN(bIsPrintingTiming);
	size_t uNrOfVoxels = uWidth * uHeight * uDepth;

	// allocate a 2D linear buffer for the time difference
	float *pfDist_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pfDist_device,
			BATCH_SIZE * sizeof(pfDist_device[0]) ) );

	float *pfDist_host;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipHostMalloc(
			&pfDist_host,
			BATCH_SIZE * sizeof(pfDist_host[0]) ) );
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	size_t uNrOfBatches = ceilf((float)uNrOfVoxels / (float)BATCH_SIZE);
	size_t uNrOfThreadsLastBatch = uNrOfVoxels % BATCH_SIZE;
	uNrOfThreadsLastBatch = (!uNrOfThreadsLastBatch)?BATCH_SIZE:uNrOfThreadsLastBatch;
	for(size_t b = 0; b < uNrOfBatches; b++) 
	{
		// allocate iNrOfElements x iNrOfTimeSteps - 1 threads
		dim3 v3Blk = dim3(BLOCK_DIM_X * BLOCK_DIM_Y);
		size_t uNrOfNeededThreads = (b == uNrOfBatches - 1)?uNrOfThreadsLastBatch:BATCH_SIZE;
		size_t uNrOfBlocks = (unsigned int)ceilf((float)uNrOfNeededThreads / (float)v3Blk.x);
		dim3 v3Grid = dim3(
			min(uNrOfBlocks, (size_t)GRID_DIM_X),
			(unsigned int)ceil((double)uNrOfBlocks / (double)GRID_DIM_X)
			);

		// invoke the kernel

		#if	IS_COMP_MULTI_POINTS	
		for(size_t p = 0; p < uNrOfPoints; p+=MAX_NR_OF_COMP_POINTS)
		{
			size_t uNrOfPointsToCompare = min(uNrOfPoints - p, (size_t)MAX_NR_OF_COMP_POINTS);
			CUDA_SAFE_CALL_NO_SYNC( 
				hipMemcpyToSymbol(HIP_SYMBOL(
					pf4Points_const),	
					&pf4Points[p],	
					sizeof(pf4Points[0]) * uNrOfPointsToCompare,
					0, hipMemcpyHostToDevice) );

			_CompDist_kernel<<<v3Grid, v3Blk, 0>>>
			(
				b * BATCH_SIZE,
				p,
				uNrOfPointsToCompare,
				uWidth,
				uHeight,
				uDepth,
				&pfDist_device[0]
			);	
			CUT_CHECK_ERROR("_CompDist_kernel() failed");
		}
		#else	// #if	IS_COMP_MULTI_POINTS	
		for(size_t p = 0; p < uNrOfPoints; p++)
		{
			_CompDist_kernel<<<v3Grid, v3Blk, 0>>>
			(
				b * BATCH_SIZE,
				p,
				pf4Points[p],
				uWidth,
				uHeight,
				uDepth,
				&pfDist_device[0]
			);	
			CUT_CHECK_ERROR("_CompDist_kernel() failed");
		}
		#endif	// #if	IS_COMP_MULTI_POINTS	

		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pfDist_host, 
				pfDist_device,
				uNrOfNeededThreads * sizeof(pfDist_host[0]),
				hipMemcpyDeviceToHost) );
		memcpy(&pfDist[b * BATCH_SIZE], &pfDist_host[0], uNrOfNeededThreads * sizeof(pfDist[0]));
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	FREE_MEMORY_ON_HOST(pfDist_host);
	FREE_MEMORY(pfDist_device);
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_PRINT(bIsPrintingTiming);
}

void
_GPUDistCompDistFromPointsToPoints
(
	size_t uNrOfPoints1,
	const float4 pf4Points1[],

	size_t uNrOfPoints2,
	const float4 pf4Points2[],

	float pfDist[],
	unsigned int puNearestPoint2[],
	void *pReserved
)
{
	if( bIsUsingCpu )
	{
		_GPUDistCompDistFromPointsToPointsByCpu
		(
			uNrOfPoints1,
			pf4Points1,

			uNrOfPoints2,
			pf4Points2,

			pfDist,
			puNearestPoint2,
			pReserved
		);
		return;
	}

LIBCLOCK_INIT(bIsPrintingTiming, __FUNCTION__);
LIBCLOCK_BEGIN(bIsPrintingTiming);
	// allocate a linear buffer for the time difference
	float4 *pf4Points1_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pf4Points1_device,
			BATCH_SIZE * sizeof(pf4Points1_device[0]) ) );

	float *pfDist_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pfDist_device,
			BATCH_SIZE * sizeof(pfDist_device[0]) ) );

	float *pfDist_host;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipHostMalloc(
			&pfDist_host,
			BATCH_SIZE * sizeof(pfDist_host[0]) ) );

	unsigned int *puNearestPoint2_device = NULL;
	unsigned int *puNearestPoint2_host = NULL;
	if( puNearestPoint2 )
	{
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMalloc(
				&puNearestPoint2_device,
				BATCH_SIZE * sizeof(puNearestPoint2_device[0]) ) );

		CUDA_SAFE_CALL_NO_SYNC( 
			hipHostMalloc(
				&puNearestPoint2_host,
				BATCH_SIZE * sizeof(puNearestPoint2_host[0]) ) );
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	size_t uNrOfBatches = (size_t)ceilf((float)uNrOfPoints1 / (float)BATCH_SIZE);
	size_t uNrOfThreadsLastBatch = uNrOfPoints1 % BATCH_SIZE;
	uNrOfThreadsLastBatch = (!uNrOfThreadsLastBatch)?BATCH_SIZE:uNrOfThreadsLastBatch;
	for(size_t b = 0; b < uNrOfBatches; b++) 
	{
		size_t uNrOfNeededThreads = (b == uNrOfBatches - 1)?uNrOfThreadsLastBatch:BATCH_SIZE;
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&pf4Points1_device[0], 
				&pf4Points1[b * BATCH_SIZE],
				uNrOfNeededThreads * sizeof(pf4Points1_device[0]),
				hipMemcpyHostToDevice) );

		// allocate iNrOfElements x iNrOfTimeSteps - 1 threads
		dim3 v3Blk = dim3(BLOCK_DIM_X * BLOCK_DIM_Y);
		size_t uNrOfBlocks = (unsigned int)ceilf((float)uNrOfNeededThreads / (float)v3Blk.x);
		dim3 v3Grid = dim3(
			min(uNrOfBlocks, (size_t)GRID_DIM_X),
			(unsigned int)ceil((double)uNrOfBlocks / (double)GRID_DIM_X)
			);

		// invoke the kernel

		#if	IS_COMP_MULTI_POINTS	
		for(size_t p2 = 0; p2 < uNrOfPoints2; p2+=MAX_NR_OF_COMP_POINTS)
		{
			size_t uNrOfPoints2ToCompare = min(uNrOfPoints2 - p2, (size_t)MAX_NR_OF_COMP_POINTS);
			CUDA_SAFE_CALL_NO_SYNC( 
				hipMemcpyToSymbol(HIP_SYMBOL(
					pf4Points_const),	
					&pf4Points2[p2],	
					sizeof(pf4Points2[0]) * uNrOfPoints2ToCompare,
					0, hipMemcpyHostToDevice) );

			_CompDistFromPoints_kernel<<<v3Grid, v3Blk, 0>>>
			(
				uNrOfNeededThreads,
				&pf4Points1_device[0],

				p2,
				uNrOfPoints2ToCompare,

				&pfDist_device[0],
				&puNearestPoint2_device[0],
				pReserved
			);	
			CUT_CHECK_ERROR("_CompDist_kernel() failed");
		}
		#else	// #if	IS_COMP_MULTI_POINTS	
		for(size_t p2 = 0; p2 < uNrOfPoints2; p2++)
		{
			_CompDistFromPoints_kernel<<<v3Grid, v3Blk, 0>>>
			(
				uNrOfNeededThreads,
				&pf4Points1_device[0],

				p2,
				pf4Points2[p2],

				&pfDist_device[0],
				&puNearestPoint2_device[0],
				pReserved
			);	
			CUT_CHECK_ERROR("_CompDist_kernel() failed");
		}
		#endif	// #if	IS_COMP_MULTI_POINTS	
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pfDist_host, 
				pfDist_device,
				uNrOfNeededThreads * sizeof(pfDist_host[0]),
				hipMemcpyDeviceToHost) );
		memcpy(&pfDist[b * BATCH_SIZE], &pfDist_host[0], uNrOfNeededThreads * sizeof(pfDist[0]));

		if(puNearestPoint2)
		{
			CUDA_SAFE_CALL_NO_SYNC( 
				hipMemcpy(
					puNearestPoint2_host, 
					puNearestPoint2_device,
					uNrOfNeededThreads * sizeof(puNearestPoint2_host[0]),
					hipMemcpyDeviceToHost) );
			memcpy(
				&puNearestPoint2[b * BATCH_SIZE], 
				&puNearestPoint2_host[0], 
				uNrOfNeededThreads * sizeof(puNearestPoint2[0]));
		}
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	FREE_MEMORY(pf4Points1_device);
	FREE_MEMORY_ON_HOST(pfDist_host);
	FREE_MEMORY(pfDist_device);
	if( puNearestPoint2 )
	{
		FREE_MEMORY_ON_HOST(puNearestPoint2_host);
		FREE_MEMORY(puNearestPoint2_device);
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_PRINT(bIsPrintingTiming);
}

void
_GPUDistCompDistFromPointsToTriangles
(
	size_t uNrOfPoints1,
	const float4 pf4Points1[],

	size_t uNrOfPoints2,
	const float4 pf4Points2[],

	size_t uNrOfTriangles,
	const ulong4 pu4TriangleVertices[],

	bool bIsPrecomputingTrasforms,

	float pfDists[]
)
{
	if( bIsUsingCpu )
	{
		_GPUDistCompDistFromPointsToTrianglesByCpu
		(
			uNrOfPoints1,
			pf4Points1,

			uNrOfPoints2,
			pf4Points2,

			uNrOfTriangles,
			pu4TriangleVertices,

			bIsPrecomputingTrasforms,

			pfDists
		);
		return;
	}

LIBCLOCK_INIT(bIsPrintingTiming, __FUNCTION__);

	TBuffer<float4> pf4Xs;
	TBuffer<float4> pf4Ys;
	TBuffer<float4> pf4Zs;
	TBuffer<float4> pf4B2s;
	TBuffer<float4> pf4C2s;
	TBuffer<float>	pfDets;
	if(bIsPrecomputingTrasforms)
	{
		LIBCLOCK_BEGIN(bIsPrintingTiming);	
		pf4Xs.alloc(uNrOfTriangles);
		pf4Ys.alloc(uNrOfTriangles);
		pf4Zs.alloc(uNrOfTriangles);
		pf4B2s.alloc(uNrOfTriangles);
		pf4C2s.alloc(uNrOfTriangles);
		pfDets.alloc(uNrOfTriangles);

		ulong4 *pu4TriangleVertices_device;	
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMalloc(
				&pu4TriangleVertices_device, 
				uNrOfTriangles * sizeof(pu4TriangleVertices_device[0]) ) );
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pu4TriangleVertices_device,	
				pu4TriangleVertices, 
				uNrOfTriangles * sizeof(pu4TriangleVertices_device[0]), hipMemcpyHostToDevice) );

		// compute the transform for all triangles
		float4 *pf4Xs_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pf4Xs_device, uNrOfTriangles * sizeof(pf4Xs_device[0]) ) );
		float4 *pf4Ys_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pf4Ys_device, uNrOfTriangles * sizeof(pf4Ys_device[0]) ) );
		float4 *pf4Zs_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pf4Zs_device, uNrOfTriangles * sizeof(pf4Zs_device[0]) ) );
		float4 *pf4B2s_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pf4B2s_device, uNrOfTriangles * sizeof(pf4B2s_device[0]) ) );
		float4 *pf4C2s_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pf4C2s_device, uNrOfTriangles * sizeof(pf4C2s_device[0]) ) );
		float *pfDets_device;	CUDA_SAFE_CALL_NO_SYNC( hipMalloc(&pfDets_device, uNrOfTriangles * sizeof(pfDets_device[0]) ) );

		////////// upload all points2 as a 2D texture to the device //////////////
		float4 *pf4Points_device;	
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMalloc(
				&pf4Points_device,
				uNrOfPoints2 * sizeof(pf4Points_device[0]) ) );
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pf4Points_device,	
				pf4Points2, 
				uNrOfPoints2 * sizeof(pf4Points_device[0]), hipMemcpyHostToDevice) );
		size_t uPointTexWidth = 4096;
		size_t uPointTexHeight = (size_t)ceil((double)uNrOfPoints2 / (double)uPointTexWidth);

		/*
		SETUP_TEXTURE(
			t2Df4Points, 
			hipAddressModeClamp, hipAddressModeClamp, hipAddressModeClamp, 
			hipFilterModePoint, false);
		*/
		t2Df4Points.normalized = false;
		t2Df4Points.filterMode = hipFilterModePoint;
		for(int a = 0; a < 3; a ++)
			t2Df4Points.addressMode[a] = hipAddressModeClamp;
		CUDA_SAFE_CALL_NO_SYNC( 
			hipBindTexture2D(
				NULL, 
				t2Df4Points, 
				&pf4Points_device[0],
				t2Df4Points.channelDesc,
				uPointTexWidth,
				uPointTexHeight,
				uPointTexWidth * sizeof(pf4Points_device[0])));

		// allocate iNrOfElements x iNrOfTimeSteps - 1 threads
		dim3 v3Blk = dim3(BLOCK_DIM_X * BLOCK_DIM_Y);
		size_t uNrOfBlocks = (unsigned int)ceilf((float)uNrOfTriangles / (float)v3Blk.x);
		dim3 v3Grid = dim3(
			min(uNrOfBlocks, (size_t)GRID_DIM_X),
			(unsigned int)ceil((double)uNrOfBlocks / (double)GRID_DIM_X)
			);

		_TransformTriangle_kernel<<<v3Grid, v3Blk, 0>>>
		(
			uNrOfTriangles,
			uPointTexWidth,
			uPointTexHeight,

			pu4TriangleVertices_device,

			pf4Xs_device,
			pf4Ys_device,
			pf4Zs_device,
			pf4B2s_device,
			pf4C2s_device,
			pfDets_device);
		CUT_CHECK_ERROR("_TransformTriangle_kernel() failed");

		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pf4Xs[0],	pf4Xs_device, uNrOfTriangles * sizeof(pf4Xs_device[0]), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pf4Ys[0],	pf4Ys_device, uNrOfTriangles * sizeof(pf4Ys_device[0]), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pf4Zs[0],	pf4Zs_device, uNrOfTriangles * sizeof(pf4Zs_device[0]), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pf4B2s[0],	pf4B2s_device, uNrOfTriangles * sizeof(pf4B2s_device[0]), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pf4C2s[0],	pf4C2s_device, uNrOfTriangles * sizeof(pf4C2s_device[0]), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL_NO_SYNC( hipMemcpy(&pfDets[0],	pfDets_device, uNrOfTriangles * sizeof(pfDets_device[0]), hipMemcpyDeviceToHost ) );

		FREE_MEMORY(pf4Xs_device);
		FREE_MEMORY(pf4Ys_device);
		FREE_MEMORY(pf4Zs_device);
		FREE_MEMORY(pf4B2s_device);
		FREE_MEMORY(pf4C2s_device);
		FREE_MEMORY(pfDets_device);
		FREE_MEMORY(pu4TriangleVertices_device);
		LIBCLOCK_END(bIsPrintingTiming);	
	}

LIBCLOCK_BEGIN(bIsPrintingTiming);
	// allocate a linear buffer for the time difference
	float4 *pf4Points1_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pf4Points1_device,
			BATCH_SIZE * sizeof(pf4Points1_device[0]) ) );

	float *pfDist_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pfDist_device,
			BATCH_SIZE * sizeof(pfDist_device[0]) ) );

	float *pfDist_host;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipHostMalloc(
			&pfDist_host,
			BATCH_SIZE * sizeof(pfDist_host[0]) ) );
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	size_t uNrOfBatches = (size_t)ceilf((float)uNrOfPoints1 / (float)BATCH_SIZE);
	size_t uNrOfThreadsLastBatch = uNrOfPoints1 % BATCH_SIZE;
	uNrOfThreadsLastBatch = (!uNrOfThreadsLastBatch)?BATCH_SIZE:uNrOfThreadsLastBatch;
	for(size_t b = 0; b < uNrOfBatches; b++) 
	{
		size_t uNrOfNeededThreads = (b == uNrOfBatches - 1)?uNrOfThreadsLastBatch:BATCH_SIZE;
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&pf4Points1_device[0], 
				&pf4Points1[b * BATCH_SIZE],
				uNrOfNeededThreads * sizeof(pf4Points1_device[0]),
				hipMemcpyHostToDevice) );

		// allocate iNrOfElements x iNrOfTimeSteps - 1 threads
		dim3 v3Blk = dim3(BLOCK_DIM_X * BLOCK_DIM_Y);
		size_t uNrOfBlocks = (unsigned int)ceilf((float)uNrOfNeededThreads / (float)v3Blk.x);
		dim3 v3Grid = dim3(
			min(uNrOfBlocks, (size_t)GRID_DIM_X),
			(unsigned int)ceil((double)uNrOfBlocks / (double)GRID_DIM_X)
			);

		// invoke the kernel
		for(size_t t = 0; t < uNrOfTriangles; t++)
		{
			///////////////////////////// compute ///////////////////////////////
			// use A as the origin
			// use BA as the Z axis
			float4 f4A = pf4Points2[pu4TriangleVertices[t].x];

			if( !bIsPrecomputingTrasforms )
			{
				float4 f4B = pf4Points2[pu4TriangleVertices[t].y];
				float4 f4C = pf4Points2[pu4TriangleVertices[t].z];

				Lib3dsVector v3X, v3Y, v3Z, v3B2, v3C2;
				float fDet;
				_CompTransform
				(
					f4A, f4B, f4C,
					v3X, v3Y, v3Z, 
					v3B2, v3C2, fDet
				 );

				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4A_const),	&f4A,	sizeof(f4A),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4B_const),	&f4B,	sizeof(f4B),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4C_const),	&f4C,	sizeof(f4C),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4X_const),	&v3X,	3 * sizeof(v3X[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4Y_const),	&v3Y,	3 * sizeof(v3Y[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4Z_const),	&v3Z,	3 * sizeof(v3Z[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4B2_const),&v3B2,	3 * sizeof(v3B2[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4C2_const),&v3C2,	3 * sizeof(v3C2[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(fDet_const),&fDet,	sizeof(fDet),	0, hipMemcpyHostToDevice) );
			}
			else
			{
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4A_const),	&f4A,		sizeof(f4A),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4X_const),	&pf4Xs[t],	sizeof(pf4Xs[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4Y_const),	&pf4Ys[t],	sizeof(pf4Ys[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4Z_const),	&pf4Zs[t],	sizeof(pf4Zs[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4B2_const),&pf4B2s[t],	sizeof(pf4B2s[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(f4C2_const),&pf4C2s[t],	sizeof(pf4C2s[0]),	0, hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL_NO_SYNC( 
					hipMemcpyToSymbol(HIP_SYMBOL(fDet_const),&pfDets[t],	sizeof(pfDets[0]),	0, hipMemcpyHostToDevice) );
			}

			///////////////////////////// invoke the kernel //////////////////////////
			_CompDistFromTriangle_kernel<<<v3Grid, v3Blk, 0>>>
			(
				uNrOfNeededThreads,
				&pf4Points1_device[0],
				t,
				&pfDist_device[0]
			);	
			CUT_CHECK_ERROR("_CompDistFromTriangle_kernel() failed");
		}
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pfDist_host, 
				pfDist_device,
				uNrOfNeededThreads * sizeof(pfDist_host[0]),
				hipMemcpyDeviceToHost) );
		memcpy(&pfDists[b * BATCH_SIZE], &pfDist_host[0], uNrOfNeededThreads * sizeof(pfDists[0]));
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	FREE_MEMORY_ON_HOST(pfDist_host);
	FREE_MEMORY(pfDist_device);
	FREE_MEMORY(pf4Points1_device);	
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_PRINT(bIsPrintingTiming);
}

void
_GPUDistCountIntersectingTriangles
(
	size_t uNrOfPoints1,
	float4 pf4Points1[],

	size_t uNrOfPoints2,
	float4 pf4Points2[],

	size_t uNrOfTriangles,
	ulong4 pu4TriangleVertices[],

	float4 f4Dir,

	float pfCount[]
)
{
LIBCLOCK_INIT(bIsPrintingTiming, __FUNCTION__);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	float4 *pf4Points1_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pf4Points1_device,
			BATCH_SIZE * sizeof(pf4Points1_device[0]) ) );

	float *pfCount_device;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMalloc(
			&pfCount_device,
			BATCH_SIZE * sizeof(pfCount_device[0]) ) );
	CUDA_SAFE_CALL_NO_SYNC( 
		hipMemset(
			pfCount_device,
			0,
			BATCH_SIZE * sizeof(pfCount_device[0]) ) );

	float *pfCount_host;
	CUDA_SAFE_CALL_NO_SYNC( 
		hipHostMalloc(
			&pfCount_host,
			BATCH_SIZE * sizeof(pfCount_host[0]) ) );

	CUDA_SAFE_CALL_NO_SYNC( 
		hipMemcpyToSymbol(HIP_SYMBOL(f4Dir_const),	&f4Dir,		sizeof(f4Dir),	0, hipMemcpyHostToDevice) );

LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	size_t uNrOfBatches = (size_t)ceilf((float)uNrOfPoints1 / (float)BATCH_SIZE);
	size_t uNrOfThreadsLastBatch = uNrOfPoints1 % BATCH_SIZE;
	uNrOfThreadsLastBatch = (!uNrOfThreadsLastBatch)?BATCH_SIZE:uNrOfThreadsLastBatch;
	for(size_t b = 0; b < uNrOfBatches; b++) 
	{
		size_t uNrOfNeededThreads = (b == uNrOfBatches - 1)?uNrOfThreadsLastBatch:BATCH_SIZE;
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&pf4Points1_device[0], 
				&pf4Points1[b * BATCH_SIZE],
				uNrOfNeededThreads * sizeof(pf4Points1_device[0]),
				hipMemcpyHostToDevice) );

		// allocate iNrOfElements x iNrOfTimeSteps - 1 threads
		dim3 v3Blk = dim3(BLOCK_DIM_X * BLOCK_DIM_Y);
		size_t uNrOfBlocks = (unsigned int)ceilf((float)uNrOfNeededThreads / (float)v3Blk.x);
		dim3 v3Grid = dim3(
			min(uNrOfBlocks, (size_t)GRID_DIM_X),
			(unsigned int)ceil((double)uNrOfBlocks / (double)GRID_DIM_X)
			);

		// invoke the kernel
		for(size_t t = 0; t < uNrOfTriangles; t++)
		{
			///////////////////////////// compute ///////////////////////////////
			// use A as the origin
			// use BA as the Z axis
			float4 f4A = pf4Points2[pu4TriangleVertices[t].x];

			float4 f4B = pf4Points2[pu4TriangleVertices[t].y];
			float4 f4C = pf4Points2[pu4TriangleVertices[t].z];

			Lib3dsVector v3X, v3Y, v3Z, v3B2, v3C2;
			float fDet;
			_CompTransform
			(
				f4A, f4B, f4C,
				v3X, v3Y, v3Z, 
				v3B2, v3C2, fDet
			 );

			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4A_const),	&f4A,	sizeof(f4A),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4X_const),	&v3X,	3 * sizeof(v3X[0]),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4Y_const),	&v3Y,	3 * sizeof(v3Y[0]),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4Z_const),	&v3Z,	3 * sizeof(v3Z[0]),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4B2_const),&v3B2,	3 * sizeof(v3B2[0]),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(f4C2_const),&v3C2,	3 * sizeof(v3C2[0]),	0, hipMemcpyHostToDevice) );
			CUDA_SAFE_CALL_NO_SYNC( hipMemcpyToSymbol(HIP_SYMBOL(fDet_const),&fDet,	sizeof(fDet),	0, hipMemcpyHostToDevice) );

			///////////////////////////// invoke the kernel //////////////////////////
			_CountIntersectingTriangle_kernel<<<v3Grid, v3Blk, 0>>>
			(
				uNrOfNeededThreads,
				&pf4Points1_device[0],
				&pfCount_device[0]
			);	
			CUT_CHECK_ERROR("_CompDistFromTriangle_kernel() failed");
		}
		CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				pfCount_host, 
				pfCount_device,
				uNrOfNeededThreads * sizeof(pfCount_host[0]),
				hipMemcpyDeviceToHost) );
		memcpy(&pfCount[b * BATCH_SIZE], &pfCount_host[0], uNrOfNeededThreads * sizeof(pfCount[0]));
	}
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_BEGIN(bIsPrintingTiming);
	FREE_MEMORY_ON_HOST(	pfCount_host		);
	FREE_MEMORY(		pfCount_device		);
	FREE_MEMORY(		pf4Points1_device	);
LIBCLOCK_END(bIsPrintingTiming);

LIBCLOCK_PRINT(bIsPrintingTiming);
}


/*

$Log$

*/